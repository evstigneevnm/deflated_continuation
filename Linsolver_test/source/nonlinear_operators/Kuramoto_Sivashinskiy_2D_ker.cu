#include "hip/hip_runtime.h"
#include <nonlinear_operators/Kuramoto_Sivashinskiy_2D_ker.h>


template<typename T, typename T_C>
__global__ void gradient_Fourier_kernel(size_t Nx, size_t My, T_C *gradient_x, T_C *gradient_y)
{

    unsigned int j=blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int k=blockDim.y * blockIdx.y + threadIdx.y;
    
    if((j>=Nx)||(k>=My)) return;

    int m=j;
    if(j>=Nx/2)
        m=j-Nx;
    
    int n=k;
    
    gradient_x[I2(j,k,Nx)]=T_C(T(0.0),T(m));
    gradient_y[I2(j,k,Nx)]=T_C(T(0.0),T(n));

}


template<typename T, typename T_C>
__global__ void Laplace_Fourier_kernel(size_t Nx, size_t My, T_C *gradient_x, T_C *gradient_y, T *Laplace)
{

    unsigned int j=blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int k=blockDim.y * blockIdx.y + threadIdx.y;
    
    if((j>=Nx)||(k>=My)) return;

    T_C x2 = gradient_x[I2(j,k,Nx)]*gradient_x[I2(j,k,Nx)];
    T_C y2 = gradient_y[I2(j,k,Nx)]*gradient_y[I2(j,k,Nx)];

    Laplace[I2(j,k,Nx)]= x2.real() + y2.real();

}

template<typename T, typename T_C>
__global__ void biharmonic_Fourier_kernel(size_t Nx, size_t My, T_C *gradient_x, T_C *gradient_y, T *biharmonic)
{


    unsigned int j=blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int k=blockDim.y * blockIdx.y + threadIdx.y;
    
    if((j>=Nx)||(k>=My)) return;

    T_C x4=gradient_x[I2(j,k,Nx)]*gradient_x[I2(j,k,Nx)]*gradient_x[I2(j,k,Nx)]*gradient_x[I2(j,k,Nx)];
    T_C y4=gradient_y[I2(j,k,Nx)]*gradient_y[I2(j,k,Nx)]*gradient_y[I2(j,k,Nx)]*gradient_y[I2(j,k,Nx)];
    T_C x2y2=gradient_x[I2(j,k,Nx)]*gradient_x[I2(j,k,Nx)]*gradient_y[I2(j,k,Nx)]*gradient_y[I2(j,k,Nx)];
    
    biharmonic[I2(j,k,Nx)]=x4.real()+T(2.0)*x2y2.real()+y4.real();

}





template<typename T, typename T_C>
void gradient_Fourier(dim3 dimGrid, dim3 dimBlock, size_t Nx, size_t My, T_C *gradient_x, T_C *gradient_y)
{
    gradient_Fourier_kernel<T,T_C><<<dimGrid, dimBlock>>>(Nx,My,gradient_x,gradient_y);
}



template<typename T, typename T_C>
void Laplace_Fourier(dim3 dimGrid, dim3 dimBlock, size_t Nx, size_t My, T_C *gradient_x, T_C *gradient_y, T *Laplce)
{
    Laplace_Fourier_kernel<T,T_C><<<dimGrid, dimBlock>>>(Nx,My,gradient_x, gradient_y, Laplce);
}



template<typename T, typename T_C>
void biharmonic_Fourier(dim3 dimGrid, dim3 dimBlock, size_t Nx, size_t My, T_C *gradient_x, T_C *gradient_y, T *biharmonic)
{
    biharmonic_Fourier_kernel<T,T_C><<<dimGrid, dimBlock>>>(Nx,My,gradient_x, gradient_y, biharmonic);
}

//explicit instantiation
template void gradient_Fourier<float, thrust::complex<float> >(dim3 dimGrid, dim3 dimBlock, size_t Nx, size_t My, thrust::complex<float> *gradient_x, thrust::complex<float> *gradient_y);
template void gradient_Fourier<double, thrust::complex<double> >(dim3 dimGrid, dim3 dimBlock, size_t Nx, size_t My, thrust::complex<double> *gradient_x, thrust::complex<double> *gradient_y);
template void Laplace_Fourier<float, thrust::complex<float> >(dim3 dimGrid, dim3 dimBlock, size_t Nx, size_t My, thrust::complex<float> *gradient_x, thrust::complex<float> *gradient_y, float *Laplce);
template void Laplace_Fourier<double, thrust::complex<double> >(dim3 dimGrid, dim3 dimBlock, size_t Nx, size_t My, thrust::complex<double> *gradient_x, thrust::complex<double> *gradient_y, double *Laplce);
template void biharmonic_Fourier<float, thrust::complex<float> >(dim3 dimGrid, dim3 dimBlock, size_t Nx, size_t My, thrust::complex<float> *gradient_x, thrust::complex<float> *gradient_y, float *biharmonic);
template void biharmonic_Fourier<double, thrust::complex<double> >(dim3 dimGrid, dim3 dimBlock, size_t Nx, size_t My, thrust::complex<double> *gradient_x, thrust::complex<double> *gradient_y, double *biharmonic);