#include "hip/hip_runtime.h"
#include <nonlinear_operators/circle/circle_ker.h>


template<typename T>
__global__ void function_kernel(int Nx, T R_, T *x, T lambda, T *f)
{

    int j=blockDim.x * blockIdx.x + threadIdx.x;
    
    if(j>=Nx) return;

    f[0] = x[0]*x[0]+lambda*lambda-R_*R_;

}


template<typename T>
__global__ void jacobian_x_kernel(int Nx, T R_, T *x0, T lambda0, T* dx, T *df)
{

    int j=blockDim.x * blockIdx.x + threadIdx.x;
    
    if(j>=Nx) return;
    // [d(x^2+lambda^2-R^2)/dx]*dx = 2x0*dx
    df[0] = T(2)*x0[0]*dx[0];

}

template<typename T>
__global__ void jacobian_lambda_kernel(int Nx, T R_, T *x0, T lambda0, T *dlambda)
{

    int j=blockDim.x * blockIdx.x + threadIdx.x;
    
    if(j>=Nx) return;

    dlambda[0] = T(2)*lambda0;

}

template<typename T>
void function(dim3 dimGrid, dim3 dimBlock, size_t Nx, const T R_, const T*& x, const T lambda, T* &f)
{
    function_kernel<T><<<dimGrid, dimBlock>>>(Nx, (T)R_,  (T*&)x, (T)lambda, f);
}


template<typename T>
void jacobian_x(dim3 dimGrid, dim3 dimBlock, size_t Nx,  const T R_, const T*& x0, const T lambda0, const T*& dx, T*& df)
{
    jacobian_x_kernel<T><<<dimGrid, dimBlock>>>(Nx, R_, (T*&) x0, (T) lambda0, (T*&) dx, (T*) df);
}

template<typename T>
void jacobian_lambda(dim3 dimGrid, dim3 dimBlock, size_t Nx, const T R_, const T*& x0, const T lambda0, T*& dlambda)
{
    jacobian_lambda_kernel<T><<<dimGrid, dimBlock>>>(Nx, R_, (T*&) x0, lambda0, dlambda);
}



//explicit instantiation
template void function<float>(dim3 dimGrid, dim3 dimBlock, size_t Nx, const float R_, const float*& x, const float lambda, float*& f);
template void jacobian_x<float>(dim3 dimGrid, dim3 dimBlock, size_t Nx, const float R_, const float*& x0, const float lambda0, const float*& dx, float*& df);
template void jacobian_lambda<float>(dim3 dimGrid, dim3 dimBlock, size_t Nx, const float R_, const float*& x0, const float lambda0, float*& dlambda);

template void function<double>(dim3 dimGrid, dim3 dimBlock, size_t Nx, const double R_, const double*& x, const double lambda, double*& f);
template void jacobian_x<double>(dim3 dimGrid, dim3 dimBlock, size_t Nx, const double R_, const double*& x0, const double lambda0, const double*& dx, double*& df);
template void jacobian_lambda<double>(dim3 dimGrid, dim3 dimBlock, size_t Nx, const double R_, const double*& x0, const double lambda0, double*& dlambda);
